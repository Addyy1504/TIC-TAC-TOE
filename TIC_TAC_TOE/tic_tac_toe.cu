#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

__device__ void make_move(char* board, int idx, char symbol) {
    if (board[idx] == ' ') board[idx] = symbol;
}

__device__ int find_winning_move(char* board, char symbol) {
    int win_positions[8][3] = {
        {0,1,2},{3,4,5},{6,7,8}, // rows
        {0,3,6},{1,4,7},{2,5,8}, // cols
        {0,4,8},{2,4,6}          // diags
    };
    for (int i = 0; i < 8; i++) {
        int a = win_positions[i][0], b = win_positions[i][1], c = win_positions[i][2];
        if (board[a] == symbol && board[b] == symbol && board[c] == ' ') return c;
        if (board[a] == symbol && board[c] == symbol && board[b] == ' ') return b;
        if (board[b] == symbol && board[c] == symbol && board[a] == ' ') return a;
    }
    return -1;
}

__global__ void playerA_turn(char* board) {
    int idx = find_winning_move(board, 'X');
    if (idx == -1) idx = find_winning_move(board, 'O'); // block O
    if (idx == -1) {
        for (int i = 0; i < 9; i++) {
            if (board[i] == ' ') {
                idx = i;
                break;
            }
        }
    }
    make_move(board, idx, 'X');
}

__global__ void playerB_turn(char* board, int seed) {
    int idx = -1;
    for (int i = 0; i < 50 && idx == -1; i++) {
        int r = (seed + i) % 9;
        if (board[r] == ' ') {
            idx = r;
        }
    }
    if (idx == -1) {
        for (int i = 0; i < 9; i++) {
            if (board[i] == ' ') {
                idx = i;
                break;
            }
        }
    }
    make_move(board, idx, 'O');
}

__host__ void print_board(const char* board) {
    for (int i = 0; i < 9; i++) {
        printf("%c ", board[i]);
        if (i % 3 == 2) printf("\n");
    }
    printf("\n");
}

__host__ void save_board(FILE* f, const char* board, int turn) {
    fprintf(f, "Turn %d:\n", turn);
    for (int i = 0; i < 9; i++) {
        fprintf(f, "%c ", board[i]);
        if (i % 3 == 2) fprintf(f, "\n");
    }
    fprintf(f, "\n");
}

__host__ bool check_win(const char* board, char symbol) {
    int win_positions[8][3] = {
        {0,1,2},{3,4,5},{6,7,8},
        {0,3,6},{1,4,7},{2,5,8},
        {0,4,8},{2,4,6}
    };
    for (int i = 0; i < 8; i++) {
        if (board[win_positions[i][0]] == symbol &&
            board[win_positions[i][1]] == symbol &&
            board[win_positions[i][2]] == symbol)
            return true;
    }
    return false;
}

__host__ bool board_full(const char* board) {
    for (int i = 0; i < 9; i++) if (board[i] == ' ') return false;
    return true;
}

int main() {
    char host_board[9];
    for (int i = 0; i < 9; i++) host_board[i] = ' ';
    char* dev_board;
    hipMalloc(&dev_board, 9 * sizeof(char));
    hipMemcpy(dev_board, host_board, 9 * sizeof(char), hipMemcpyHostToDevice);

    FILE* f = fopen("replay.txt", "w");

    int turn = 0;
    srand(time(NULL));
    while (true) {
        if (turn % 2 == 0) {
            playerA_turn<<<1, 1>>>(dev_board);
        } else {
            int seed = rand();
            playerB_turn<<<1, 1>>>(dev_board, seed);
        }

        hipMemcpy(host_board, dev_board, 9 * sizeof(char), hipMemcpyDeviceToHost);
        print_board(host_board);
        save_board(f, host_board, turn);

        if (check_win(host_board, 'X')) {
            fprintf(f, "Player A (X) wins!\n");
            printf("Player A (X) wins!\n");
            break;
        } else if (check_win(host_board, 'O')) {
            fprintf(f, "Player B (O) wins!\n");
            printf("Player B (O) wins!\n");
            break;
        } else if (board_full(host_board)) {
            fprintf(f, "It's a draw!\n");
            printf("It's a draw!\n");
            break;
        }

        turn++;
    }

    fclose(f);
    hipFree(dev_board);
    return 0;
}
